// /opt/cuda/bin/nvcc main.cu -o main


#include <hip/hip_runtime.h>
#include <cstdint>
#include <stdint.h>
#include <stdio.h>

#define MODULE_N 256
#define CUDA_BLOCK_N 4096

__global__ void rand_uintN(uint8_t *r) { *r = blockIdx.x % MODULE_N; }

void rand_uintNs(uint8_t *gamma, int n);
void print_uintNs(uint8_t *gamma, int n);
void write_uintNs(uint8_t *gamma, int n);
void print_uintNs_count(uint8_t *gamma, int n);

int main(int argc, char *argv[]) {
  const int n = 4096;
  uint8_t gamma[n];
  
  rand_uintNs(gamma, n);
  write_uintNs(gamma, n);
  return 0;
}

void rand_uintNs(uint8_t *gamma, int n) {
  int num_count = n * MODULE_N;

  uint8_t raw_rand[num_count];
  uint8_t *dev_r;

  memset(raw_rand, 0, sizeof(raw_rand));
  hipMalloc(&dev_r, sizeof(uint8_t));
  for (int i = 0; i < num_count; i++) {
    rand_uintN<<<CUDA_BLOCK_N, 1>>>(dev_r);
    hipMemcpy(raw_rand + i, dev_r, sizeof(uint8_t), hipMemcpyDeviceToHost);
  }
  hipFree(dev_r);

  for (int i = 0; i < num_count; i += MODULE_N) {
    int sum = 0;
    for (int j = 0; j < MODULE_N; ++j) {
      sum += raw_rand[i + j];
    }
    gamma[i / MODULE_N] = sum % MODULE_N;
  }
}

void write_uintNs(uint8_t *gamma, int n) {
  fwrite(gamma, sizeof(uint8_t), n, stdout);
}
